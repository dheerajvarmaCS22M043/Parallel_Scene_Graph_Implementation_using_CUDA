#include "hip/hip_runtime.h"

/*
	CS 6023 Assignment 3. 
	Do not make any changes to the boiler plate code or the other files in the folder.
	Use hipFree to deallocate any memory not in usage.
	Optimize as much as possible.
 */

#include "SceneNode.h"
#include <queue>
#include "Renderer.h"
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <chrono>


void readFile (const char *fileName, std::vector<SceneNode*> &scenes, std::vector<std::vector<int> > &edges, std::vector<std::vector<int> > &translations, int &frameSizeX, int &frameSizeY) {
	/* Function for parsing input file*/

	FILE *inputFile = NULL;
	// Read the file for input. 
	if ((inputFile = fopen (fileName, "r")) == NULL) {
		printf ("Failed at opening the file %s\n", fileName) ;
		return ;
	}

	// Input the header information.
	int numMeshes ;
	fscanf (inputFile, "%d", &numMeshes) ;
	fscanf (inputFile, "%d %d", &frameSizeX, &frameSizeY) ;
	

	// Input all meshes and store them inside a vector.
	int meshX, meshY ;
	int globalPositionX, globalPositionY; // top left corner of the matrix.
	int opacity ;
	int* currMesh ;
	for (int i=0; i<numMeshes; i++) {
		fscanf (inputFile, "%d %d", &meshX, &meshY) ;
		fscanf (inputFile, "%d %d", &globalPositionX, &globalPositionY) ;
		fscanf (inputFile, "%d", &opacity) ;
		currMesh = (int*) malloc (sizeof (int) * meshX * meshY) ;
		for (int j=0; j<meshX; j++) {
			for (int k=0; k<meshY; k++) {
				fscanf (inputFile, "%d", &currMesh[j*meshY+k]) ;
			}
		}
		//Create a Scene out of the mesh.
		SceneNode* scene = new SceneNode (i, currMesh, meshX, meshY, globalPositionX, globalPositionY, opacity) ; 
		scenes.push_back (scene) ;
	}

	// Input all relations and store them in edges.
	int relations;
	fscanf (inputFile, "%d", &relations) ;
	int u, v ; 
	for (int i=0; i<relations; i++) {
		fscanf (inputFile, "%d %d", &u, &v) ;
		edges.push_back ({u,v}) ;
	}

	// Input all translations.
	int numTranslations ;
	fscanf (inputFile, "%d", &numTranslations) ;
	std::vector<int> command (3, 0) ;
	for (int i=0; i<numTranslations; i++) {
		fscanf (inputFile, "%d %d %d", &command[0], &command[1], &command[2]) ;
		translations.push_back (command) ;
	}
}


void writeFile (const char* outputFileName, int *hFinalPng, int frameSizeX, int frameSizeY) {
	/* Function for writing the final png into a file.*/
	FILE *outputFile = NULL; 
	if ((outputFile = fopen (outputFileName, "w")) == NULL) {
		printf ("Failed while opening output file\n") ;
	}
	
	for (int i=0; i<frameSizeX; i++) {
		for (int j=0; j<frameSizeY; j++) {
			fprintf (outputFile, "%d ", hFinalPng[i*frameSizeY+j]) ;
		}
		fprintf (outputFile, "\n") ;
	}
}


__global__ void bfs(int* dOffset, int* dCsr,  int* dVisited, bool* dFinished,  int* dx,  int* dy, int V, int E){
    unsigned  int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < V){
        if(dVisited[id] == 1){ //node is active
            
            for(int nbrId = dOffset[id]; nbrId < dOffset[id+1]; ++nbrId){
                int nbr = dCsr[nbrId];
                if(dVisited[nbr] == 0){
                    dVisited[nbr] = 1;
                    dFinished[0] = false;
                }
                // printf("%d  %d\n", dx[nbr], dy[nbr]);
                // printf("dheeraj\n");
                atomicAdd(&dx[nbr], dx[id]);
                atomicAdd(&dy[nbr], dy[id]);
                // printf("%d %d\n", dx[nbr], dy[nbr]);
            }
            dVisited[id] =  2;
        }
    }
}

__global__ void newXY(int* x1, int* y1, int* x2, int* y2, int* dx, int* dy, int V){
    unsigned  int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < V){
        x2[id] = x1[id] + dx[id];
        y2[id] = y1[id] + dy[id];
    }
}

__global__ void calOpacity(int* ddOpacity, int* dFrameSizeX, int* dFrameSizeY, int V, int* x, int* y, int* dOpacity, int frameSizeX, int frameSizeY){
    unsigned  int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < V){
        for(int len = 0; len < dFrameSizeX[id]; len++){
            for(int wid = 0; wid < dFrameSizeY[id]; wid++){
                int tempX, tempY;
                tempX = x[id] + len;
                tempY = y[id] + wid;
                if(tempX >=0 && tempX < frameSizeX && tempY >= 0 && tempY < frameSizeY)
                 atomicMax(&(ddOpacity[tempX * frameSizeY + tempY]), dOpacity[id]);
            }
        }
    }
}

__global__ void calMeshValue(int* dMesh, int *dMeshOffset, int* ddOpacity, int* dOpacity, int* dFrameSizeX, int* dFrameSizeY, int V, int* x, int* y, int frameSizeX, int frameSizeY, int* dFinalPng){
    unsigned  int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < V){
        for(int len = 0; len < dFrameSizeX[id]; len++){
            for(int wid = 0; wid < dFrameSizeY[id]; wid++){
                int tempX, tempY;
                tempX = x[id] + len;
                tempY = y[id] + wid;
                // printf("Mesh id = %d, x = %d, y = %d, tempX = %d, tempY = %d, dOpacity = %d, cellOpacity = %d, width = %d, height = %d\n", id, x[id], y[id], tempX, tempY, dOpacity[id], ddOpacity[tempX * frameSizeY + tempY], dFrameSizeX[id], dFrameSizeY[id]);
                if(((tempX >=0 && tempX < frameSizeX && tempY >= 0 && tempY < frameSizeY)) && (ddOpacity[tempX * frameSizeY + tempY] == dOpacity[id])) {
                    
                    int meshX = len, meshY = wid;
                        int offset = dMeshOffset[id];
                        dFinalPng[tempX * frameSizeY + tempY] = dMesh[offset + meshX*dFrameSizeY[id] + meshY];
                        
                }
            }
        }
        
    }

}


int main (int argc, char **argv) {
	
	// Read the scenes into memory from File.
	const char *inputFileName = argv[1] ;
	int* hFinalPng ; 

	int frameSizeX, frameSizeY ;
	std::vector<SceneNode*> scenes ;
	std::vector<std::vector<int> > edges ;
	std::vector<std::vector<int> > translations ;
	readFile (inputFileName, scenes, edges, translations, frameSizeX, frameSizeY) ;
	hFinalPng = (int*) malloc (sizeof (int) * frameSizeX * frameSizeY) ;
	
	// Make the scene graph from the matrices.
    Renderer* scene = new Renderer(scenes, edges) ;

	// Basic information.
	int V = scenes.size () ;
	int E = edges.size () ;
	int numTranslations = translations.size () ;

	// Convert the scene graph into a csr.
	scene->make_csr () ; // Returns the Compressed Sparse Row representation for the graph.
	int *hOffset = scene->get_h_offset () ;  
	int *hCsr = scene->get_h_csr () ;
	int *hOpacity = scene->get_opacity () ; // hOpacity[vertexNumber] contains opacity of vertex vertexNumber.
	int **hMesh = scene->get_mesh_csr () ; // hMesh[vertexNumber] contains the mesh attached to vertex vertexNumber.
	int *hGlobalCoordinatesX = scene->getGlobalCoordinatesX () ; // hGlobalCoordinatesX[vertexNumber] contains the X coordinate of the vertex vertexNumber.
	int *hGlobalCoordinatesY = scene->getGlobalCoordinatesY () ; // hGlobalCoordinatesY[vertexNumber] contains the Y coordinate of the vertex vertexNumber.
	int *hFrameSizeX = scene->getFrameSizeX () ; // hFrameSizeX[vertexNumber] contains the vertical size of the mesh attached to vertex vertexNumber.
	int *hFrameSizeY = scene->getFrameSizeY () ; // hFrameSizeY[vertexNumber] contains the horizontal size of the mesh attached to vertex vertexNumber.

	auto start = std::chrono::high_resolution_clock::now () ;


	// Code begins here.
	// Do not change anything above this comment.
    
    //step1:
    //dx and dy arrays for translation purpose
     int* x = new  int[V];
     int* y = new  int[V];

    //initializing the dx and dy arrays with 0's 
    for(int i=0;i<V;i++){
        x[i] = 0;
        y[i] = 0;
    }

    //now applying the translations individually:
    for(int i=0;i<numTranslations;i++){
        int n = translations[i][0], c = translations[i][1], a = translations[i][2];
        if(c == 0){
            x[n] -= a;
        }
        else if(c == 1){
            x[n] += a;
        }
        else if(c == 2){
            y[n] -= a;
        }
        else{
            y[n] += a;
        }
        
    }

    // for(int i=0;i<V;i++){
    //     printf("%d  %d\n",x[i], y[i]);
    // }
    // printf("varma\n");


    //step 2:
     int* visited = new  int[V];
    for(int i=0;i<V;i++){
        visited[i] = 0;
    }
    visited[0] = 1;

     int* dVisited; //gpu

    hipMalloc(&dVisited, sizeof( int)*(V));
    hipMemcpy(dVisited, visited, sizeof( int)*(V), hipMemcpyHostToDevice);

    int* dOffset; //gpu
    int* dCsr;
    hipMalloc(&dOffset, sizeof(int)*(V+1));
    hipMalloc(&dCsr, sizeof(int)*(E));
    hipMemcpy(dOffset, hOffset, sizeof(int)*(V+1), hipMemcpyHostToDevice);
    hipMemcpy(dCsr, hCsr, sizeof(int)*(E), hipMemcpyHostToDevice);

     int* dx; //gpu
     int* dy;
    hipMalloc(&dx, sizeof( int)*(V));
    hipMalloc(&dy, sizeof( int)*(V));
    hipMemcpy(dx, x, sizeof(int)*(V), hipMemcpyHostToDevice);
    hipMemcpy(dy, y, sizeof(int)*(V), hipMemcpyHostToDevice);

    int nBlocks = (V+1023)/(1024);

    while(1){
        bool isFinished[1];
        isFinished[0] = true;
        bool* dFinished; //gpu
        hipMalloc(&dFinished, sizeof(bool));
        hipMemcpy(dFinished, isFinished, sizeof(bool), hipMemcpyHostToDevice);
        
        bfs<<<nBlocks, 1024>>>(dOffset, dCsr, dVisited, dFinished, dx, dy, V, E);
        hipDeviceSynchronize();

        hipMemcpy(isFinished, dFinished, sizeof(bool), hipMemcpyDeviceToHost);
        hipFree(dFinished);
        if(isFinished[0] == true) break;   
        
    }

    hipFree(dVisited);
    hipFree(dCsr);
    hipFree(dOffset);

    hipMemcpy(x,dx, sizeof(int)*(V), hipMemcpyDeviceToHost);
    hipMemcpy(y,dy, sizeof(int)*(V), hipMemcpyDeviceToHost);
    // for(int i=0;i<V;i++){
    //     printf("%d  %d\n",x[i], y[i]);
    // }

    //part 3:
     int* x1; //gpu
     int* y1;
    // for(int i=0;i<V;i++){
    //     printf("%d %d\n", hGlobalCoordinatesX[i], hGlobalCoordinatesY[i]);
    // }
    hipMalloc(&x1, sizeof( int)*(V));
    hipMalloc(&y1, sizeof( int)*(V));
    hipMemcpy(x1, hGlobalCoordinatesX, sizeof(int)*(V), hipMemcpyHostToDevice);
    hipMemcpy(y1, hGlobalCoordinatesY, sizeof(int)*(V), hipMemcpyHostToDevice);
     int* x2, *y2; //gpu
    hipMalloc(&x2, sizeof( int)*(V));
    hipMalloc(&y2, sizeof( int)*(V));

    newXY<<<nBlocks, 1024>>>(x1, y1, x2, y2, dx, dy, V);  //kernel for calculating new x,y
    hipDeviceSynchronize();

    // hipMemcpy(hGlobalCoordinatesX, x2, sizeof( int)*(V), hipMemcpyHostToDevice);
    // hipMemcpy(hGlobalCoordinatesY, y2, sizeof( int)*(V), hipMemcpyHostToDevice);

    hipFree(x1);
    hipFree(y1);
    // hipFree(x2);
    // hipFree(y2);

    //cuda dx, dy, x2, y2 are alive

    int* ddOpacity;
    hipMalloc(&ddOpacity, sizeof(int)*(frameSizeX)*(frameSizeY));
    int val = INT_MIN;
    hipMemset(ddOpacity, val, sizeof(int)*(frameSizeX)*(frameSizeY)); //initialization 

    int* dFrameSizeX, *dFrameSizeY;
    hipMalloc(&dFrameSizeX, sizeof(int)*(V));
    hipMalloc(&dFrameSizeY, sizeof(int)*(V));
    hipMemcpy(dFrameSizeX, hFrameSizeX, sizeof(int)*(V), hipMemcpyHostToDevice);
    hipMemcpy(dFrameSizeY, hFrameSizeY, sizeof(int)*(V), hipMemcpyHostToDevice);

    int* dOpacity;
    hipMalloc(&dOpacity, sizeof(int)*(V));
    hipMemcpy(dOpacity, hOpacity, sizeof(int)*(V), hipMemcpyHostToDevice);

    calOpacity<<<nBlocks, 1024>>>(ddOpacity, dFrameSizeX, dFrameSizeY, V, x2, y2, dOpacity, frameSizeX, frameSizeY); //kernel to calculate max Opacity value of each pixel
    hipDeviceSynchronize();

    

    //hipMemcpy(hFinalPng, ddOpacity, sizeof(int)*(frameSizeX)*(frameSizeY), hipMemcpyDeviceToHost);

    // int** dMesh;
    // hipMalloc(&dMesh, sizeof(int*)*V);
    // for(int i=0;i<V;i++){
    //     int sizeI = hFrameSizeX[i] * hFrameSizeY[i];
    //     int *dTemp;
    //     hipMalloc(&dTemp, sizeof(int)*(sizeI));
    //     hipMemcpy(dTemp, hMesh[i], sizeof(int)*(sizeI), hipMemcpyHostToDevice);
    //     hipMemcpy(&dMesh[i] , &dTemp, sizeof(int*) ,hipMemcpyDeviceToDevice);
    //     hipFree(dTemp);
        
    // }

    int *dMesh, *dMeshOffset;
    int *hMeshOffset = (int*) malloc (sizeof (int) * (V+1));

    int sum = 0;
    hMeshOffset[0] = 0;
    for(int i=1;i<=V;i++) {
        sum += hFrameSizeX[i-1] * hFrameSizeY[i-1];
        hMeshOffset[i] = sum;
    }
    hipMalloc(&dMesh, sizeof (int) * sum);
    sum = 0;
    for(int i=0;i<V;i++) {
        
        hipMemcpy(dMesh + sum, hMesh[i], sizeof(int)*(hFrameSizeX[i] * hFrameSizeY[i]), hipMemcpyHostToDevice);
        sum += hFrameSizeX[i] * hFrameSizeY[i];
    }
    hipMalloc(&dMeshOffset, sizeof (int) * (V+1));
    hipMemcpy(dMeshOffset, hMeshOffset, sizeof(int)*(V+1), hipMemcpyHostToDevice);

    int* dFinalPng; //gpu
    hipMalloc(&dFinalPng, sizeof (int) * frameSizeX * frameSizeY);
    hipMemset(dFinalPng, 0, sizeof(int)*(frameSizeX)*(frameSizeY));

    // for(int i=0;i<V;i++){

    //     int* dMesh;
    //     hipMalloc(&dMesh, sizeof(int)*(hFrameSizeX[i] * hFrameSizeY[i]));
    //     hipMemcpy(dMesh, hMesh[i], sizeof(int)*(hFrameSizeX[i] * hFrameSizeY[i]), hipMemcpyHostToDevice);

        
    // }

    calMeshValue<<<nBlocks, 1024>>>(dMesh, dMeshOffset, ddOpacity, dOpacity, dFrameSizeX, dFrameSizeY, V, x2, y2, frameSizeX, frameSizeY, dFinalPng);
    hipDeviceSynchronize();

    hipMemcpy(hFinalPng, dFinalPng, sizeof(int)*(frameSizeX)*(frameSizeY), hipMemcpyDeviceToHost);

    hipFree(dMesh);
    hipFree(ddOpacity);
    hipFree(dFrameSizeX);
    hipFree(dFrameSizeY);
    hipFree(dx);
    hipFree(dFinalPng);





	// Do not change anything below this comment.
	// Code ends here.

	auto end  = std::chrono::high_resolution_clock::now () ;

	std::chrono::duration<double, std::micro> timeTaken = end-start;

	printf ("execution time : %f\n", timeTaken) ;
	// Write output matrix to file.
	const char *outputFileName = argv[2] ;
	writeFile (outputFileName, hFinalPng, frameSizeX, frameSizeY) ;	

}

